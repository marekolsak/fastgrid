#include "hip/hip_runtime.h"
/*
    AutoGrid

    Copyright (C) 1989-2007, Garrett M. Morris, David S. Goodsell, Ruth Huey, Arthur J. Olson,
    All Rights Reserved.
    Copyright (C) 2008-2009, Marek Olsak (maraeo@gmail.com), All Rights Reserved.

    AutoGrid is a Trade Mark of The Scripps Research Institute.

    This program is free software; you can redistribute it and/or
    modify it under the terms of the GNU General Public License
    as published by the Free Software Foundation; either version 2
    of the License, or (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program; if not, write to the Free Software
    Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/

#include "../autogrid.h"
#include "cuda_internal.h"
#include <cstdio>

// Grid size and spacing
static __constant__ int2 numGridPointsDiv2;
static __constant__ int numGridPointsX;
static __constant__ float gridSpacing;

// Per-slice parameters
static __constant__ int outputIndexZBase, numAtoms;
static __constant__ float4 atoms[NUM_ATOMS_PER_KERNEL]; // {x, y, (z-gridPosZ)^2, charge}

// Generic kernel
template<int DistanceDependentDielectric>
static __global__ void calcGridPoint(float *outEnergies, const float *epsilon)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float gridPosX = (x - numGridPointsDiv2.x) * gridSpacing;
    float gridPosY = (y - numGridPointsDiv2.y) * gridSpacing;

    float energy = 0;

    //  Do all Receptor (protein, DNA, etc.) atoms...
    for (int ia = 0; ia < numAtoms; ia++)
    {
        // Get the distance from current grid point to this receptor atom (|receptorAtom - gridPos|)
        float dx = atoms[ia].x - gridPosX;
        float dy = atoms[ia].y - gridPosY;
        float rSq = dx*dx + dy*dy + atoms[ia].z;
        float invR = rsqrt(rSq);

        // The estat forcefield coefficient/weight is premultiplied
        if (DistanceDependentDielectric)
            energy += atoms[ia].w * min(invR, 2.f) * epsilon[min(int(A_DIVISOR / invR), int(MAX_DIST-1))];
        else
            energy += atoms[ia].w * min(invR, 2.f);
    }

    // Round to 3 decimal places
    int outputIndex = outputIndexZBase + y * numGridPointsX + x;
    outEnergies[outputIndex] += energy;
}

void setGridMapParametersAsyncCUDA(const int *numGridPointsX, const int2 *numGridPointsDiv2XY, const float *gridSpacing, hipStream_t stream)
{
    // Set common variables
    CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL("numGridPointsDiv2"), numGridPointsDiv2XY, sizeof(int2),  0, hipMemcpyHostToDevice, stream));
    CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL("numGridPointsX"),    numGridPointsX,      sizeof(int),   0, hipMemcpyHostToDevice, stream));
    CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL("gridSpacing"),       gridSpacing,         sizeof(float), 0, hipMemcpyHostToDevice, stream));
}

void setGridMapSliceParametersAsyncCUDA(const int *outputIndexZBase, hipStream_t stream)
{
    CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL("outputIndexZBase"), outputIndexZBase, sizeof(int), 0, hipMemcpyHostToDevice, stream));
}

void setGridMapKernelParametersAsyncCUDA(const int *numAtoms, const float4 *atoms, hipStream_t stream)
{
    CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL("atoms"),    atoms,    sizeof(float4) * *numAtoms, 0, hipMemcpyHostToDevice, stream));
    CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL("numAtoms"), numAtoms, sizeof(int),                0, hipMemcpyHostToDevice, stream));
}

void callKernelAsyncCUDA(const dim3 &grid, const dim3 &block, float *outEnergies, const float *epsilon, hipStream_t stream)
{
    if (epsilon)
        CUDA_SAFE_KERNEL((calcGridPoint<1><<<grid, block, stream>>>(outEnergies, epsilon)));
    else
        CUDA_SAFE_KERNEL((calcGridPoint<0><<<grid, block, stream>>>(outEnergies, 0)));
}

void checkErrorCUDA(hipError_t e, const char *file, int line, const char *func, const char *code)
{
    if (e != hipSuccess)
        fprintf(stderr, "CUDA error: '%s'\n"
                        "        in file '%s'\n"
                        "        in line %i\n"
                        "        in function '%s'\n"
                        "        in code '%s'\n", hipGetErrorString(e), file, line, func, code);
}
